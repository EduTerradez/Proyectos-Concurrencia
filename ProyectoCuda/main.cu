#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
using namespace std;

//poner la \ en cada linea menos en el ultimo parentesis
#define cudaCheckError(){ \
cudaError_t e = hipGetLastError(); \
if (e != hipSuccess){ \
	printf("Cuda failures %s: %d: '%s'\n", __FILE__,__LINE__, hipGetErrorString(e)); \
	exit(0); \
	} \
}

__device__ float multiplicarVector(int lado, float* fila, float* columna) {
	float resultado = 0;
	for (int i = 0; i < lado; i++) {
		resultado += fila[i] * columna[i];
	}
	return resultado;
}

__global__ void multiplicarMatrices_kernel(int size, float** matriz1, float** matriz2, float** resultado) {
	int col = blockIdx.y * blockDim.x + threadIdx.x;
	int fil = blockIdx.x * blockDim.y + threadIdx.y;

	if((col >= size) || (fil>= size)){
		return;
	}
	//printf("fila: %d columna: %d dato1: %f dato2: %f \n", fil, col, matriz1[fil][col], matriz2[col][fil]);
	resultado[fil][col] = multiplicarVector(size, matriz1[fil], matriz2[col]);

}

__global__ void verContenido(float** resultado) {
	int col = blockIdx.y * blockDim.x + threadIdx.x;
	int fil = blockIdx.x * blockDim.y + threadIdx.y;
	printf("dato: %f \n", resultado[fil][col]);
}

float** leerMatriz(char* nombreFichero, bool traspuesta, int size){
		ifstream fichero(nombreFichero,std::fstream::in | std::fstream::binary);
		fichero.read((char*)&nombreFichero,sizeof(int));

		float** matriz = (float**)malloc(size*sizeof(float*));
		int i;
		for(i = 0; i < size;i++){
			matriz[i] = (float*)malloc(size*sizeof(float));
		}

		if(!traspuesta){

			for(int i = 0; i < size;i++){
				for(int j = 0; j < size;j++){
					float aux;
					fichero.read((char*)&aux,sizeof(float));
					matriz[i][j] = aux;
				}
			}
		}
		else{
			for(int i = 0; i < size;i++){
							for(int j = 0; j < size;j++){
								float aux;
								fichero.read((char*)&aux,sizeof(float));
								matriz[j][i] = aux;
							}
						}

		}
		fichero.close();
		return matriz;
}

void pasarMatrizFichero(int lado, char* nombreFichero, float** matriz) {
	fstream fichero(nombreFichero, std::fstream::out | std::fstream::binary);

	char straux[100];


	sprintf(straux, "%d ", lado);

	fichero.write(straux, strlen(straux));

	float aux;
	for (int i = 0; i < lado; i++) {
		sprintf(straux, "\n");
		fichero.write(straux, strlen(straux));
		for (int j = 0; j < lado; j++) {
			aux = matriz[i][j];


			sprintf(straux, "%f ", aux);
			fichero.write(straux, strlen(straux));
		}
	}
	fichero.close();
}

int main(int argc, char** argv) {
	int SIZE = 10;

	if (argc > 1) {
		SIZE = atoi(argv[1]);
	} else {
		printf("Error en los parametros\n");
	}
	cout << SIZE << endl;
	float** matriz1_CPU;
	float** matriz2_CPU;
	float** resultadoFinal_CPU;

	float** datosM1_CPU;
	float** datosM2_CPU;
	float **resultado_CPU;

	float** resultado_GPU;
	float** datosM1_GPU;
	float** datosM2_GPU;




	//obtener matrices de fichero
	matriz1_CPU = leerMatriz("matriz1", false, SIZE);
	matriz2_CPU =leerMatriz("matriz2", true, SIZE);

/*

	for(int i = 0; i<SIZE;i++){
		for(int j = 0; j < SIZE;j++){
			cout << " "<< matriz1_CPU[i][j];
		}
		cout << endl;
	}

	for(int i = 0; i<SIZE;i++){
			for(int j = 0; j < SIZE;j++){
				if(i == j){
					matriz2_CPU[i][j] = 1;
				}
				cout << " "<< matriz2_CPU[i][j];
			}
			cout << endl;
		}
*/
	//funsiona

	//reserva de memoria del resultado
	resultadoFinal_CPU= (float**)malloc(SIZE * sizeof(float*));
	for(int i = 0; i < SIZE;i++){
		resultadoFinal_CPU[i] = (float*)malloc(SIZE*sizeof(float));
	}

	datosM1_CPU = (float**)malloc(SIZE * sizeof(float*));
	datosM2_CPU = (float**)malloc(SIZE * sizeof(float*));
	resultado_CPU = (float**)malloc(SIZE * sizeof(float*));

	hipMalloc((void**)&resultado_GPU,sizeof(float*)* SIZE);
	hipMalloc((void**)&datosM2_GPU,sizeof(float*)* SIZE);
	hipMalloc((void**)&datosM1_GPU,sizeof(float*)* SIZE);

	for(int i = 0; i < SIZE; i++){
		hipMalloc((void**)&(datosM1_CPU[i]),sizeof(float)* SIZE);
		hipMalloc((void**)&(datosM2_CPU[i]),sizeof(float)* SIZE);
		hipMalloc((void**)&(resultado_CPU[i]),sizeof(float)* SIZE);
	}

	for(int i = 0; i < SIZE;i++){
		hipMemcpy(datosM1_CPU[i], matriz1_CPU[i], SIZE * sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(datosM2_CPU[i], matriz2_CPU[i], SIZE * sizeof(float),hipMemcpyHostToDevice);
	}

	hipMemcpy(datosM1_GPU, datosM1_CPU, SIZE * sizeof(float*),hipMemcpyHostToDevice);
	hipMemcpy(datosM2_GPU, datosM2_CPU, SIZE * sizeof(float*),hipMemcpyHostToDevice);
	hipMemcpy(resultado_GPU, resultado_CPU, SIZE * sizeof(float*),hipMemcpyHostToDevice);

	int blockSize = 32;

	dim3 gridDim = dim3((SIZE / blockSize) + 1, (SIZE / blockSize) + 1, 1);
	dim3 blockDim = dim3(blockSize, blockSize, 1);

	multiplicarMatrices_kernel <<<gridDim, blockDim>>> (SIZE, datosM1_GPU, datosM2_GPU, resultado_GPU);


	for(int i = 0; i < SIZE;i++){
		hipMemcpy(resultadoFinal_CPU[i], resultado_CPU[i], SIZE * sizeof(float),hipMemcpyDeviceToHost);
	}

	pasarMatrizFichero(SIZE, "MatrizResultado", resultadoFinal_CPU);

	//free

	for(int i = 0; i < SIZE; i++){
		free(matriz1_CPU[i]);
		free(matriz2_CPU[i]);
		free(resultadoFinal_CPU[i]);

		hipFree(datosM1_CPU[i]);
		hipFree(datosM2_CPU[i]);

		hipFree(resultado_CPU[i]);

	}

	free(matriz1_CPU);
	free(matriz2_CPU);
	free(resultadoFinal_CPU);
	free(resultado_CPU);
	free(datosM1_CPU);
	free(datosM2_CPU);


	hipFree(resultado_GPU);
	hipFree(datosM1_GPU);
	hipFree(datosM2_GPU);

	//free(resultadoAuxiliar_CPU);

}
